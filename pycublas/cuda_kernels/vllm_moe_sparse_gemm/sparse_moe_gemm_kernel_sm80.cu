#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <torch/extension.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"


template <class ProblemShape, class CtaTiler,
          class TA, class AStride, class ASmemLayout, class TiledCopyA,
          class TB, class BStride, class BSmemLayout, class TiledCopyB,
          class TC, class CStride, class CSmemLayout, class TiledMma,
          class Alpha, class Beta>
__global__ static
__launch_bounds__(decltype(size(TiledMma{}))::value)
void
sparse_gemm_device(ProblemShape shape_MNK, CtaTiler cta_tiler,
                  TA const* A, AStride dA, ASmemLayout sA_layout, TiledCopyA copy_a,
                  TB const* B, BStride dB, BSmemLayout sB_layout, TiledCopyB copy_b,
                  TC      * C, CStride dC, CSmemLayout          , TiledMma mma,
                  Alpha alpha, Beta beta, TA* b_scale, TA* topk_weight,
                  int32_t* sorted_token_ids, int32_t* expert_ids,
                  int32_t* num_tokens_post_padded, int32_t num_valid_tokens)
{
  using namespace cute;
  Tensor mA = make_tensor(make_gmem_ptr(A), select<0,2>(shape_MNK), dA); // (M,K)
  Tensor mB = make_tensor(make_gmem_ptr(B), select<1,2>(shape_MNK), dB); // (N,K)
  Tensor mC = make_tensor(make_gmem_ptr(C), select<0,1>(shape_MNK), dC); // (M,N)
}

// Setup params for a NT GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
sparse_gemm_nt(int m, int n, int k,
              Alpha alpha,
              TA const* A, int ldA,
              TB const* B, int ldB,
              Beta beta,
              TC      * C, int ldC,
              hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define NT strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);                      // (dM, dK)
  auto dB = make_stride(Int<1>{}, ldB);                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int<  8>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<3>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA = make_layout(make_shape(bM, bK, bP));             // (m,k,p) -> smem_idx; m-major
  auto sB = make_layout(make_shape(bN, bK, bP));             // (n,k,p) -> smem_idx; n-major
  auto sC = make_layout(make_shape(bM, bN));                 // (m,n) -> smem_idx; m-major

  // Define the thread layouts (static)

  TiledCopy copyA = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint64_t>, TA>{},
                                    Layout<Shape<_32,_8>>{}, // Thr layout 32x8 m-major
                                    Layout<Shape< _4,_1>>{});// Val layout  4x1 m-major
  TiledCopy copyB = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint32_t>, TB>{},
                                    Layout<Shape<_32,_8>>{}, // Thr layout 32x8 n-major
                                    Layout<Shape< _4,_1>>{});// Val layout  4x1 n-major

  TiledMMA mmaC = make_tiled_mma(UniversalFMA<TC,TA,TB>{},
                                 Layout<Shape<_16,_16,_1>>{});  // 16x16x1 TiledMMA

#if 0
  print(copyA);
  print(copyB);
  print(mmaC);
#endif

#if 0
  print_latex(copyA);
  print_latex(copyB);
  print_latex(mmaC);
#endif

  dim3 dimBlock(size(mmaC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  sparse_gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, copyA,
       B, dB, sB, copyB,
       C, dC, sC, mmaC,
       alpha, beta, nullptr,
       nullptr, nullptr, nullptr,
       nullptr, 0
      );
}


template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
sparse_gemm(char transA, char transB, int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  if (transA == 'N' && transB == 'T') {
    return sparse_gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  }
  assert(false && "Not implemented");
}

/*
int main(int argc, char** argv)
{
  hipDeviceProp_t props;
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major < 8) {
    std::cout << "This example requires an Ampere GPU or newer (CC >= 80)" << std::endl;
    // Return 0 so tests pass if run on unsupported architectures or CUDA Toolkits.
    return 0;
  }

  int m = 5120;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 5120;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 4096;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  char transA = 'N';
  if (argc >= 5)
    sscanf(argv[4], "%c", &transA);

  char transB = 'T';
  if (argc >= 6)
    sscanf(argv[5], "%c", &transB);

  using TA = half;
  using TB = uint8_t;
  using TC = float;
  using TI = float;

  TI alpha = 1.0;
  TI beta  = 0.0;

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;
  std::cout << "C = A^" << transA << " B^" << transB << std::endl;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>(8); //static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<TC>(-1);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  double gflops = (2.0*m*n*k) * 1e-9;

  const int timing_iterations = 100;
  GPU_Clock timer;

  int ldA = 0, ldB = 0, ldC = m;

  if (transA == 'N') {
    ldA = m;
  } else if (transA == 'T') {
    ldA = k;
  } else {
    assert(false);
  }

  if (transB == 'N') {
    ldB = k;
  } else if (transB == 'T') {
    ldB = n;
  } else {
    assert(false);
  }

  // Run once
  d_C = h_C;
  sparse_gemm(transA, transB, m, n, k,
              alpha,
              d_A.data().get(), ldA,
              d_B.data().get(), ldB,
              beta,
              d_C.data().get(), ldC);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    sparse_gemm(transA, transB, m, n, k,
                alpha,
                d_A.data().get(), ldA,
                d_B.data().get(), ldB,
                beta,
                d_C.data().get(), ldC);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);

  return 0;
}
*/

namespace cuda_kernels{
void vllm_sparse_moe_gemm_kernel(
    torch::Tensor activation,
    torch::Tensor weight,
    torch::Tensor w_scales,
    torch::Tensor topk_weight,
    torch::Tensor sorted_token_ids,
    torch::Tensor expert_ids,
    torch::Tensor num_tokens_post_padded,
    int num_valid_tokens,
    int block_m_size,
    int64_t stream)
{
  TORCH_CHECK(activation.is_cuda(), "activation must be a CUDA tensor");
  TORCH_CHECK(weight.is_cuda(), "weight must be a CUDA tensor");
  TORCH_CHECK(w_scales.is_cuda(), "w_scales must be a CUDA tensor");
  TORCH_CHECK(topk_weight.is_cuda(), "topk_weight must be a CUDA tensor");
  TORCH_CHECK(sorted_token_ids.is_cuda(), "sorted_token_ids must be a CUDA tensor");
  TORCH_CHECK(expert_ids.is_cuda(), "expert_ids must be a CUDA tensor");
  TORCH_CHECK(num_tokens_post_padded.is_cuda(), "num_tokens_post_padded must be a CUDA tensor");

  TORCH_CHECK(activation.dtype() == torch::kHalf, "activation must be half");
  TORCH_CHECK(weight.dtype() == torch::kUInt8, "weight must be fp8(uint8)");
  TORCH_CHECK(w_scales.dtype() == torch::kHalf, "w_scales must be half");
  TORCH_CHECK(topk_weight.dtype() == torch::kHalf, "topk_weight must be half");
  TORCH_CHECK(sorted_token_ids.dtype() == torch::kInt32, "sorted_token_ids must be int32");
  TORCH_CHECK(expert_ids.dtype() == torch::kInt32, "expert_ids must be int32");
  TORCH_CHECK(num_tokens_post_padded.dtype() == torch::kInt32, "num_tokens_post_padded must be int32");

  auto M = activation.size(0);
  auto H = activation.size(1);
  auto I = weight.size(1);
  auto E = weight.size(0);

  TORCH_CHECK(H == weight.size(2), "weight shape mismatch");
}
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
  m.def(vllm_sparse_moe_gemm_kernel, &cuda_kernels::vllm_sparse_moe_gemm_kernel, "");
}